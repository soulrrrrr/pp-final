#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "md5.h"

#define MESSAGE_LENGTH 8
#define LIMIT 100000000
#define THREAD_COUNT 1024

int main(int argc, char **argv) {

    // initiate answer
    uint8_t *ans;
    hipMallocManaged((void**)&ans, sizeof(uint8_t) * 16);
    for (int i = 0; i < 16; i++) {
        char a = argv[1][i * 2];
        char b = argv[1][i * 2 + 1];
        a = (a <= '9') ? a - '0' : (a & 0x7) + 9;
        b = (b <= '9') ? b - '0' : (b & 0x7) + 9;
        ans[i] = (a << 4) + b;
    }

    // uint8_t *d_ans;
    // hipHostMalloc(&d_ans, 16*sizeof(uint8_t));
    // hipMemcpy(d_ans, ans, 16*sizeof(uint8_t), hipMemcpyHostToDevice);
    int *val;
    hipMallocManaged((void**)&val, sizeof(int) * 1); 
    // hipMalloc(&d_val, 1*sizeof(int));
    int block = (LIMIT / THREAD_COUNT) + 1;
    md5<<<block, THREAD_COUNT>>>(MESSAGE_LENGTH, ans, val);
    hipDeviceSynchronize();
    // hipMemcpy(&val, d_val, 1*sizeof(int), hipMemcpyDeviceToHost);

    printf("The number is %08d\n", *val);

    return 0;
}